#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <limits.h>

extern "C"
{
#include "initializers.h"
#include "activations.cuh"
#include "utils.h"
}

void activation_present()
{
    printf("Activation Present\n");
}

__device__ float sigmoid(float x)
{
    return 1.0 / (1.0 + exp(-x));
}

__global__ void sigmoid_kernel(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = sigmoid(input[idx]);
    }
}

extern "C" tensor sigmoid_activation(tensor rw)
{
    int total_data = rw.size[0] * rw.size[1];
    unsigned int num_threads = NUM_THREADS;
    unsigned int num_blocks = ceil((float)total_data / num_threads);

    float *dvector, *dres_vector, *vector = convert2DTo1D(rw.matrix, rw.size[0], rw.size[1]);
    hipMalloc(&dvector, total_data * sizeof(float));
    hipMalloc(&dres_vector, total_data * sizeof(float));

    hipMemcpy(dvector, vector, total_data * sizeof(float), hipMemcpyHostToDevice);
    sigmoid_kernel<<<num_blocks, num_threads>>>(dvector, dres_vector, total_data);
    hipMemcpy(vector, dres_vector, total_data * sizeof(float), hipMemcpyDeviceToHost);

    rw.matrix = convert1DTo2D(vector, rw.size[0], rw.size[1]);

    hipFree(dvector);
    hipFree(dres_vector);
    return rw;
}

__device__ float relu(float x)
{
    return MAX(0.0, x);
}

__global__ void relu_kernel(float *input, float *output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        output[idx] = relu(input[idx]);
    }
}

extern "C" tensor relu_activation(tensor rw)
{
    int total_data = rw.size[0] * rw.size[1];
    unsigned int num_threads = 1024;
    unsigned int num_blocks = ceil((float)total_data / num_threads);

    float *dvector, *dres_vector, *vector = convert2DTo1D(rw.matrix, rw.size[0], rw.size[1]);
    hipMalloc(&dvector, total_data * sizeof(float));
    hipMalloc(&dres_vector, total_data * sizeof(float));

    hipMemcpy(dvector, vector, total_data * sizeof(float), hipMemcpyHostToDevice);
    relu_kernel<<<num_blocks, num_threads>>>(dvector, dres_vector, total_data);
    hipMemcpy(vector, dres_vector, total_data * sizeof(float), hipMemcpyDeviceToHost);

    rw.matrix = convert1DTo2D(vector, rw.size[0], rw.size[1]);

    hipFree(dvector);
    hipFree(dres_vector);
    return rw;
}