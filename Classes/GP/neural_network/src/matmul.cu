#include "hip/hip_runtime.h"
// kernel.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

extern "C"
{
#include "matmul.cuh"
}
__global__ void matmul_present_kernel()
{
    printf("Matmul present\n");
}

extern "C" void matmul_present()
{
    matmul_present_kernel<<<1, 1>>>();
    hipDeviceSynchronize();
}
