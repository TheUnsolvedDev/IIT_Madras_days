#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

void save_array(const char *filename, int *arr, int length)
{
    FILE *file = fopen(filename, "wb");
    if (file == NULL)
    {
        printf("Error opening file %s for writing.\n", filename);
        return;
    }
    fwrite(arr, sizeof(int), length, file);
    fclose(file);
}

int *read_array(const char *filename, int length)
{
    FILE *file = fopen(filename, "rb");
    if (file == NULL)
    {
        printf("Error opening file %s for reading.\n", filename);
        return NULL;
    }
    fseek(file, 0, SEEK_END);
    length = ftell(file) / sizeof(int);
    rewind(file);
    int *arr = (int *)malloc((length) * sizeof(int));

    if (arr == NULL)
    {
        printf("Memory allocation failed.\n");
        fclose(file);
        return NULL;
    }
    fread(arr, sizeof(int), length, file);
    fclose(file);

    return arr;
}

__global__ void add_chain(int *z, int *x, int *y)
{
    unsigned int id = threadIdx.x;
    z[id] = pow(x[id], 2) + pow(y[id], 3);
}

void print_list(int *array, int a_length)
{
    for (int i = 0; i < a_length; i++)
        printf("%d \t", array[i]);
    printf("\n");
}

int main()
{
    int N = 100;

    int *x = (int *)malloc(N * sizeof(int));
    int *y = (int *)malloc(N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        x[i] = i % (N / 10);
        y[i] = i % (N / 20);
    }
    save_array("x.bin", x, N);
    save_array("y.bin", y, N);

    int *new_x = read_array("x.bin", N);
    int *new_y = read_array("y.bin", N);

    int z[N], *d_z, *d_y, *d_x;
    hipMalloc(&d_x, N * sizeof(int));
    hipMalloc(&d_y, N * sizeof(int));
    hipMalloc(&d_z, N * sizeof(int));

    hipMemcpy(d_x, new_x, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, new_y, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, N * sizeof(int), hipMemcpyHostToDevice);

    add_chain<<<1, N>>>(d_z, d_x, d_y);
    hipDeviceSynchronize();
    hipMemcpy(x, d_x, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, N * sizeof(int), hipMemcpyDeviceToHost);
    print_list(x, N);
    print_list(y, N);
    print_list(z, N);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    free(x);
    free(y);
    free(new_x);
    free(new_y);
    return 0;
}