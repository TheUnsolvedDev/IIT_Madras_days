#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10



__global__ void array_fill(int *array)
{
    array[threadIdx.x] = threadIdx.x * threadIdx.x;
}

void print_array(int *array, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%d \t", array[i]);
    }
    printf("\n");
}

int main()
{
    int a[N] = {0}, *d_a;
    print_array(a, N);

    hipMalloc(&d_a, sizeof(int) * N);
    array_fill<<<1, N>>>(d_a);
    hipMemcpy(a, d_a, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipFree(d_a);
    print_array(a, N); // cuda device synchronise not needed

    // cudaDeviceSynchronize();
    return 0;
}