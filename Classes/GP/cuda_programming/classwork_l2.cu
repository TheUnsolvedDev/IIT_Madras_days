#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10

__global__ void print_square()
{
    for (int i = 0; i < N; i++)
    {
        printf("%d \t", i * i);
    }
    printf("\n");
}

__global__ void thread_print()
{
    printf("%d \n", threadIdx.x);
}

__global__ void array_fill(int *array)
{
    array[threadIdx.x] = threadIdx.x * threadIdx.x;
}

void print_array(int *array, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%d \t", array[i]);
    }
    printf("\n");
}


int main()
{
    // print_square<<<1, 1>>>();
    // thread_print<<<1, N>>>();

    int a[N] = {0}, *d_a;
    print_array(a, N);

    hipMalloc(&d_a, sizeof(int) * N);
    array_fill<<<1, N>>>(d_a);
    hipMemcpy(a, d_a, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipFree(d_a);
    print_array(a, N); // cuda device synchronise not needed

    // cudaDeviceSynchronize();
    return 0;
}