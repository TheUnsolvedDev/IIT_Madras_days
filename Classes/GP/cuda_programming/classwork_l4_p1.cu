#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void init(int *array, int a_length)
{
    unsigned int id = threadIdx.x;
    if (id < a_length)
    {
        array[id] = 0;
    }
}

__global__ void add(int *array, int a_length)
{
    unsigned id = threadIdx.x;
    if (id < a_length)
    {
        array[id] += id;
    }
}

void print_list(int *array, int a_length)
{
    for (int i = 0; i < a_length; i++)
        printf("%d \t", array[i]);
    printf("\n");
}


int main()
{
    int *d_a;
    int n = 1024;
    hipMalloc(&d_a, n * sizeof(int));

    init<<<1, n>>>(d_a, n);
    add<<<1, n>>>(d_a, n);
    hipDeviceSynchronize();

    int array[n];
    hipMemcpy(array, d_a, n * sizeof(int), hipMemcpyDeviceToHost);

    print_list(array, n);
    hipFree(d_a);

    return 0;
}