#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}

int main()
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc(&d_a, size);
	hipMalloc(&d_b, size);
	hipMalloc(&d_c, size);

	a = 2, b = 102;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1, 1>>>(d_a, d_b, d_c);
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	printf("The result is %d\n", c);

	return 0;
}
