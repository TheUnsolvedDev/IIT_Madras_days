/**
 *   CS6023: GPU Programming
 *   Assignment 1
 *
 *   Please don't change any existing code in this file.
 *
 *   You can add your code whereever needed. Please add necessary memory APIs
 *   for your implementation. Use cudaFree() to free up memory as soon as you're
 *   done with an allocation. This will ensure that you don't run out of memory
 *   while running large test cases. Use the minimum required memory for your
 *   implementation. DO NOT change the kernel configuration parameters.
 */

#include <chrono>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

using std::cin;
using std::cout;

__global__ void CalculateHadamardProduct(long int *A, long int *B, int N)
{
    // TODO: Write your kernel here
    unsigned int index = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;

    // unsigned int index = threadIdx.x + blockDim.x + blockIdx.x;
    if (index < N * N)
        A[index] = A[index] * B[(index % N) * N + (index / N)];
}

__global__ void FindWeightMatrix(long int *A, long int *B, int N)
{
    // TODO: Write your kernel here
    unsigned int index = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;
    if (index < N * N)
        A[index] = max(A[index], B[index]);
}

__global__ void CalculateFinalMatrix(long int *A, long int *B, int N)
{

    // TODO: Write your kernel here
    unsigned int index = (threadIdx.x + blockDim.x * blockIdx.x) + (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;

    if (index < 4 * N * N)
    {
        int row = index / (2 * N);
        int col = index % (2 * N);

        int row_offset, col_offset;
        if (row >= N)
            row_offset = (row - N);
        else
            row_offset = row;

        if (col >= N)
            col_offset = (col - N);
        else
            col_offset = col;
        B[row * 2 * N + col] *= A[row_offset * N + col_offset];
    }
}

int main(int argc, char **argv)
{

    int N;
    cin >> N;
    long int *A = new long int[N * N];
    long int *B = new long int[N * N];
    long int *C = new long int[N * N];
    long int *D = new long int[2 * N * 2 * N];

    for (long int i = 0; i < N * N; i++)
    {
        cin >> A[i];
    }

    for (long int i = 0; i < N * N; i++)
    {
        cin >> B[i];
    }

    for (long int i = 0; i < N * N; i++)
    {
        cin >> C[i];
    }

    for (long int i = 0; i < 2 * N * 2 * N; i++)
    {
        cin >> D[i];
    }

    /**
     *
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     *
     */

    long int *d_A;
    long int *d_B;
    long int *d_C;
    long int *d_D;

    hipMalloc(&d_A, N * N * sizeof(long int));
    hipMalloc(&d_B, N * N * sizeof(long int));
    hipMalloc(&d_C, N * N * sizeof(long int));
    hipMalloc(&d_D, 4 * N * N * sizeof(long int));

    hipMemcpy(d_A, A, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, N * N * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_D, D, 4 * N * N * sizeof(long int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(1024, 1, 1);
    dim3 blocksPerGrid(ceil(N * N / 1024.0), 1, 1);

    auto start = std::chrono::high_resolution_clock::now();
    CalculateHadamardProduct<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed1 = end - start;

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(N * N / 1024.0), 1, 1);

    start = std::chrono::high_resolution_clock::now();
    FindWeightMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_C, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed2 = end - start;

    threadsPerBlock = dim3(32, 32, 1);
    blocksPerGrid = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);

    start = std::chrono::high_resolution_clock::now();
    CalculateFinalMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_D, N);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed3 = end - start;

    // Make sure your final output from the device is stored in d_D.

    /**
     *
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     *
     */

    hipMemcpy(D, d_D, 2 * N * 2 * N * sizeof(long int), hipMemcpyDeviceToHost);

    std::ofstream file("cuda.out");
    if (file.is_open())
    {
        for (long int i = 0; i < 2 * N; i++)
        {
            for (long int j = 0; j < 2 * N; j++)
            {
                file << D[i * 2 * N + j] << " ";
            }
            file << "\n";
        }
        file.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if (file2.is_open())
    {
        file2 << elapsed1.count() << "\n";
        file2 << elapsed2.count() << "\n";
        file2 << elapsed3.count() << "\n";
        file2.close();
    }
    else
    {
        std::cout << "Unable to open file";
    }

    return 0;
}