#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define MAX(x, y) (((x) > (y)) ? (x) : (y))

__global__ void trm()
{
    printf("%d %d %d\t", threadIdx.x, threadIdx.y, blockIdx.x);
}

__host__ void print_list(int *a, int N)
{
    for (int i = 0; i < N; i++)
    {
        printf("%d\t", a[i]);
        if ((i + 1) % (int)sqrt(N) == 0)
            printf("\n");
    }
    printf("\n");
}

__host__ void max_cpu(int *a, int *b, int *c, int N)
{
    for (int i = 0; i < N; i++)
    {
        c[i] = MAX(a[i], b[i]);
    }
}

__host__ void check_correct(int *a, int *b, int N)
{
    for (int i = 0; i < N; i++)
    {
        if (a[i] != b[i])
        {
            printf("Failed!\n");
            return;
        }
    }
    printf("Passed\n");
}

__global__ void max_gpu(int *a, int *b, int *c, int N)
{

    int idx = (threadIdx.x * blockDim.x + threadIdx.y) * gridDim.x + blockIdx.x;
    if (idx <= N)
    {
        c[idx] = MAX(a[idx], b[idx]);
        // printf(" %d %d \t", c[idx], MAX(a[idx], b[idx]));
    }
}

__host__ void hadamard_quad_cpu(int *a, int *b, int *d, int N)
{
    int d_size = 4 * N * N;
    int quad1 = 0, quad2 = N, quad3 = 2 * N * N, quad4 = N + (2 * N * N);
    for (int i = 0; i < N * N; i++)
    {
        d[i + quad1] = i;
        d[i + quad2] = i;
        d[i + quad3] = i;
        d[i + quad4] = i;
        if ((i + 1) % N == 0)
        {
            quad1 += N;
            quad2 += N;
            quad3 += N;
            quad4 += N;
        }
    }
}

__global__ void hadamard_quad_gpu(int *a, int *b, int *d, int N)
{
    // int idx
}

int main()
{
    int N = 10;
    int NN = N * N;
    printf("NN: %d\n", NN);

    int *a = (int *)malloc(NN * sizeof(int));
    int *b = (int *)malloc(NN * sizeof(int));
    int *c_cpu = (int *)malloc(NN * sizeof(int));
    int *c_gpu = (int *)malloc(NN * sizeof(int));
    int *d = (int *)malloc(4 * NN * sizeof(int));
    int *e_cpu = (int *)malloc(4 * NN * sizeof(int));
    int *e_gpu = (int *)malloc(4 * NN * sizeof(int));

    for (int i = 0; i < NN; i++)
    {
        a[i] = (((i + 1) * 2) - ((i + 1) % 2)) % N;
        b[i] = (((i + 1) * 2) - (i % 2)) % N;
        c_cpu[i] = 0;
        c_gpu[i] = 0;
    }
    for (int i = 0; i < 4 * NN; i++)
    {
        d[i] = 0;
        e_cpu[i] = 0;
        e_gpu[i] = 0;
    }

    max_cpu(a, b, c_cpu, NN);

    int *da, *db, *dc_gpu, *dd, *de_gpu;
    hipMalloc(&da, NN * sizeof(int));
    hipMalloc(&db, NN * sizeof(int));
    hipMalloc(&dc_gpu, NN * sizeof(int));
    hipMalloc(&dd, 4 * NN * sizeof(int));
    hipMalloc(&de_gpu, 4 * NN * sizeof(int));

    hipMemcpy(da, a, NN * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, NN * sizeof(int), hipMemcpyHostToDevice);

    dim3 threads(32, 32, 1);
    dim3 blocks(ceil(NN / 1024.0), 1, 1);
    max_gpu<<<blocks, threads>>>(da, db, dc_gpu, NN);
    hipDeviceSynchronize();

    hipMemcpy(c_gpu, dc_gpu, NN * sizeof(int), hipMemcpyDeviceToHost);
    print_list(c_cpu, NN);
    check_correct(c_cpu, c_gpu, NN);

    // threads = dim3(32, 32, 1);
    // blocks = dim3(ceil(2 * N / 32.0), ceil(2 * N / 32.0), 1);
    hadamard_quad_cpu(a, d, e_cpu, N);
    print_list(e_cpu, 4 * NN);

    hipFree(da);
    hipFree(db);
    hipFree(dc_gpu);
    hipFree(dd);
    hipFree(de_gpu);

    free(a);
    free(b);
    free(c_cpu);
    free(c_gpu);
    free(d);
    free(e_cpu);
    free(e_gpu);

    printf("\n");
    return 0;
}