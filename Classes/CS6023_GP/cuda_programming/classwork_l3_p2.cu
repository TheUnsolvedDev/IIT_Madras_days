#include <stdio.h>
#include <hip/hip_runtime.h>
#include <strings.h>

__global__ void assign(char *array, int a_length)
{
    unsigned int id = threadIdx.x;
    if (id < a_length)
    {
        ++array[id];
    }
}


int main()
{
    char cpu_arr[] = "akdjdjaskda";
    char *gpu_arr;
    hipMalloc(&gpu_arr, (1 + strlen(cpu_arr)) * sizeof(char));
    hipMemcpy(gpu_arr, cpu_arr, (1 + strlen(cpu_arr)) * sizeof(char), hipMemcpyHostToDevice);

    assign<<<1, 32>>>(gpu_arr, strlen(cpu_arr));
    hipDeviceSynchronize();
    hipMemcpy(cpu_arr, gpu_arr, (1 + strlen(cpu_arr)) * sizeof(char), hipMemcpyDeviceToHost);
    printf("%s \n",cpu_arr);

    hipFree(gpu_arr);

    return 0;
}